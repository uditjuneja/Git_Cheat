#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void add(int x, int y, int* sum){
	*sum = x + y;
	printf("BlockID: %d\tThread ID: %d\tSum is %d\n",blockIdx.x,threadIdx.x,*sum);
}

int main(){
	int a, b, sum=0;
	int *sumd;
	cout<<"\nEnter A: "; cin>>a;
	cout<<"\nEnter B: "; cin>>b;
	
	hipMalloc(&sumd, sizeof(int));
	hipMemcpy(sumd, &sum, sizeof(int), hipMemcpyHostToDevice);
	add<<<5,2>>>(a, b, sumd);
	hipMemcpy(&sum, sumd, sizeof(int), hipMemcpyDeviceToHost);
	cout<<"\nSum is "<<sum<<endl;
	return 0;
}
