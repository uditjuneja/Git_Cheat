#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

// #define N 10

__global__ void addPair(int *dev_vec, int *N)
{
	int index = 2 * threadIdx.x;

	if (index + 1  < *N)
	{
		int sum = dev_vec[index] + dev_vec[index + 1];
		// cout << "A[" << index << "] + A[" << index + 1 << "] = " << sum << endl;
		printf("A[%d] + A[%d] = %d\n", index, index + 1, sum );
	}
}

int main()
{
	int *vec, *dev_vec, *n_c, N = 10;
	
	int size = N * sizeof(int);
	
	vec = (int*)malloc(size);

	cout << "Enter 10 elements";
	
	for(int i = 0; i < N; i++)
	{
		cin >> vec[i];
	}

	hipMalloc(&dev_vec, size);

	hipMalloc(&n_c, sizeof(int));
	
	hipMemcpy(dev_vec, vec, size, hipMemcpyHostToDevice);

	hipMemcpy(n_c, &N, sizeof(int), hipMemcpyHostToDevice);	

	addPair<<<5,2>>>(dev_vec, n_c);

	hipMemcpy(vec, dev_vec, size, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	return 0;
}