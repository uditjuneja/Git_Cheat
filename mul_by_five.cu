#include<iostream>
#include<hip/hip_runtime.h>

#define N 10

using namespace std;

__global__ void mul(int* a_d, int n){
	// printf("%d %d %d\n", blockIdx.x,blockDim.x,threadIdx.x);
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < n){
		a_d[index] *= 5; 
	}
}

int main(){
	hipEvent_t start, stop;

  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	int *a, *a_d;
	
	int size = N * sizeof(int);
	
	a = (int*) malloc(size);
	
	cout << "Enter " << N << " numbers: "; 
	for(int i=0; i<N; i++){
		cin>>a[i];
	}
	
	hipMalloc(&a_d, size);
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
	
	hipEventRecord(start);
	mul<<<1,10>>>(a_d,N);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	hipMemcpy(a, a_d, size, hipMemcpyDeviceToHost);

	cout<<"Matrix After Multiplying:\n";
	for(int i=0; i<N; i++){
		cout<<a[i]<<" ";
	}

	float millis = 0;
  	hipEventElapsedTime(&millis, start, stop);
  	cout << "\nElasped Time: " << millis << endl;

	return 0;
}
