#include<iostream>
#include<hip/hip_runtime.h>


using namespace std;

__global__ void colonel(int *a_d){
	//std::cout<<"\nHello Cuda"; //cannot be used in global device
	// *a_d = 2;
	printf("\nblockIdx.x: %d\tblockIdx.y: %d\ttheradIdx.x: %d\tthreadIdx.y: %d",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);

	// cout << "Block ID - " << blockIdx.x << " " << blockIdx.y << " " << blockIdx.z << "\	n";
	// cout << "Thread ID - " << threadIdx.x << " " << threadIdx.y << " " << threadIdx.z << "\n";

}

int main(){
	int a = 0, *a_d;
	hipMalloc((void**) &a_d, sizeof(int));
	hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

	dim3 grid(2, 1, 0);
	dim3 block(1, 2, 0);

	colonel<<<grid, block>>>(a_d);

	hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

	cout<<"\na = "<<a<<endl;
	hipFree(a_d);
}
