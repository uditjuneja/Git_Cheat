#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void func(int *dev_arr1, int *dev_arr2, int *dev_dot){
  __shared__ int temp[3];
  int index = threadIdx.x;
  if (index < 3){
    temp[index] = dev_arr1[index] * dev_arr2[index];
  }
  __syncthreads();
  if (index == 0){
    *dev_dot = temp[0] + temp[1] + temp[2];
  }
}

int main(){
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *arr1, *arr2, *dot;
  int *dev_arr1, *dev_arr2, *dev_dot;

  int size = 3 * sizeof(int); // i j k
  
  arr1 = (int*) malloc(size);
  arr2 = (int*) malloc(size);
  dot = (int*) malloc(sizeof(int));

  hipMalloc(&dev_arr1, size);
  hipMalloc(&dev_arr2, size);
  hipMalloc(&dev_dot, sizeof(int));


  cout << "Enter values of x, y, z. Vector1: xi + yj + zk: ";
  cin>>arr1[0]>>arr1[1]>>arr1[2];
  cout << "Enter values of x, y, z. Vector2: xi + yj + zk: ";
  cin>>arr2[0]>>arr2[1]>>arr2[2];


  hipMemcpy(dev_arr1, arr1, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_arr2, arr2, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_dot, dot, sizeof(int), hipMemcpyHostToDevice);


  hipEventRecord(start);
  func<<<1,3>>>(dev_arr1, dev_arr2, dev_dot);
  hipEventRecord(stop);

  hipMemcpy(dot, dev_dot, sizeof(int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  cout << "Dot product is: " << *dot << endl;

  float millis = 0;
  hipEventElapsedTime(&millis, start, stop);
  cout << "Elasped Time: " << millis << endl; 
  return 0;
}