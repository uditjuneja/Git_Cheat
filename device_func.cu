#include<iostream>
#include<hip/hip_runtime.h>


using namespace std;

__device__ void print(int a, int b, int sum){
	printf("Printing sum inside DEVICE\n");
	printf("\n%d\t%d + %d = %d", threadIdx.x, a, b, sum);
}

__global__ void add(int *a, int *b, int* sum){
	*sum = *a + *b;
	print(*a, *b, *sum);
}

int main(){
	int *a, *b, *sum;
	hipMallocManaged(&a, sizeof(int));
	hipMallocManaged(&b, sizeof(int));
	hipMallocManaged(&sum , sizeof(int));
	cout<<"Enter A: ";	cin>>*a;
	cout<<"Enter B: ";	cin>>*b;
	add<<<1,10>>>(a,b,sum);
	hipDeviceSynchronize();
	cout<<"\nPrinting sum in HOST: Sum is "<<*sum<<endl;
	return 0;
}
