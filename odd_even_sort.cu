#include<iostream>
#include<hip/hip_runtime.h>


using namespace std;
#define N 10

#define intswap(A,B) {int temp=A;A=B;B=temp;}

__global__ void sort(int *c,int *count)
{
    int l;
    if(*count%2==0)
          l=*count/2;
    else
         l=(*count/2)+1;
    for(int i=0;i<l;i++)
    {
            if((!(threadIdx.x&1)) && (threadIdx.x<(*count-1)))  //even phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }

            __syncthreads();
            if((threadIdx.x&1) && (threadIdx.x<(*count-1)))     //odd phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }
            __syncthreads();
    }//for

}



int main()
{int a[N],b[N],n;
    printf("enter size of array");
    scanf("%d",&n);
    if (n > N) {printf("too large!\n"); return 1;}
    printf("enter the elements of array");
  for(int i=0;i<n;i++)
  {
    scanf("%d",&a[i]);
  }
  printf("ORIGINAL ARRAY : \n");
  for(int i=0;i<n;i++)
          {

          printf("%d ",a[i]);
          }
  int *c,*count;
  hipMalloc((void**)&c,sizeof(int)*N);
  hipMalloc((void**)&count,sizeof(int));
  hipMemcpy(c,&a,sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(count,&n,sizeof(int),hipMemcpyHostToDevice);
  sort<<< 1,n >>>(c,count);
  hipMemcpy(&b,c,sizeof(int)*N,hipMemcpyDeviceToHost);
  printf("\nSORTED ARRAY : \n");
  for(int i=0;i<n;i++)
      {
         printf("%d ",b[i]);
      }

  printf("\n");
}
