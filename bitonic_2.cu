#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 8
#define intswap(A,B) {int temp=A;A=B;B=temp;}
__global__ void sort(int *c)
{
 __shared__ int shared[N];
 int i = threadIdx.x;
 shared[i] = c[i];
 __syncthreads();
 for(int k=2;k<=8;k*=2){
 for(int j=k/2;j>0;j/=2){
 int xorres = i^j;

 if(xorres>i){
 if((i&k) == 0){
 if(shared[i]>shared[xorres])
 intswap(shared[i],shared[xorres]);
 }

 else{
 if(shared[i]<shared[xorres])
 intswap(shared[i],shared[xorres]);
 }
 }

 __syncthreads();
 }
 }
 c[i] = shared[i];
}
int main(){
 int a[N] = {6,1,2,5,3,4,7,9};
 int b[N];
 int n = N;
 printf("ORIGINAL ARRAY : \n");
 for(int i=0;i<n;i++)
 printf("%d ",a[i]);
 int *c;
 hipMalloc((void**)&c,sizeof(int)*N);
 hipMemcpy(c,&a,sizeof(int)*N,hipMemcpyHostToDevice);
 sort<<< 1,N >>>(c);
10
 hipMemcpy(b,c,sizeof(int)*N,hipMemcpyDeviceToHost);
 printf("\nSORTED ARRAY : \n");
 for(int i=0;i<N;i++)
 printf("%d ",b[i]);
 printf("\n");
 hipFree(c);
}
